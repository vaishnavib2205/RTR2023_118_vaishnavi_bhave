#include "hip/hip_runtime.h"

#include "SineWave.cu.h"

// cuda kernel for sine wave

__global__ void sineWaveKernel(float4 *pos , int width , int height , float time)
{
    // code 
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = (float)x / (float)width;
    float v = (float)y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    float frequency = 4.0f;
	float w = sinf(u * frequency + time) * cosf(v * frequency + time) * 0.5f;

    pos[y * width + x] = make_float4(u , w , v , 1.0f);

}

// user define functon to call cuda kernel

void launchCUDAKernel(float4 *pos , int width , int height , float time)
{
    dim3 block(8,8,1);
    dim3 grid(width / block.x , height / block.y , 1);

    sineWaveKernel<<<grid , block>>>(pos , width , height , time);
    
}
