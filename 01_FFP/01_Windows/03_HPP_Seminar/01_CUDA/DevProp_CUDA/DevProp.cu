#include<stdio.h>

int main(void)
{
  //function declarations
  void PrintCUDADeviceProperties(void);

  //code
  PrintCUDADeviceProperties();
}

void PrintCUDADeviceProperties(void)
{
  //code
  printf("CUDA INFORMATION: \n");
  printf("===================================================================================================\n");

  hipError_t ret_cuda_rt;
  int dev_count;
  ret_cuda_rt = hipGetDeviceCount(&dev_count);
  if(ret_cuda_rt != hipSuccess)
  {
    printf("CUDA Runtime API Error - hipGetDeviceCount() failed due to %s \n", hipGetErrorString(ret_cuda_rt));
  } 
  else if(dev_count == 0)
  {
    printf("There is NO CUDA supported device on this system. \n");
    return;
  }else
  {
   printf("Total number of CUDA supporting GPU Device/ Device on this system : %d\n", dev_count);
   for(int i = 0; i< dev_count; i++)
   {
     hipDeviceProp_t dev_prop;
     int driverVersion = 0, runtimeVersion = 0;

     ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
     if(ret_cuda_rt != hipSuccess)
     {
       printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
       return;
     }
     printf("\n");
     hipDriverGetVersion(&driverVersion);
     hipRuntimeGetVersion(&runtimeVersion);

     printf("******************** CUDA DRIVER AND RUNTIME INFORMATION *******************\n");
     printf("=============================================================================\n");
     printf("CUDA Driver Version                    : %d.%d\n", driverVersion / 1000, (driverVersion % 100) /10);
     printf("CUDA Runtime Version                    : %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) /10);
     printf("=============================================================================\n");
     printf("********************* GPU DEVICE GENERAL INFORMATION ************************ \n");
     printf("==============================================================================\n");
     printf("GPU DEVICE NUMBER                                                  :%d\n", i);
     printf("GPU DEVICE NUMBER                                                  :%s\n", dev_prop.name);
     printf("GPU Device Compute Capability                                      :%d.%d\n", dev_prop.major, dev_prop.minor);
     printf("GPU Device Clock Rate                                              :%d\n", dev_prop.clockRate);
     printf("GPU Device Type                                                    : ");
     if(dev_prop.integrated)
          printf("Integrated (On-Board)\n");
     else
          printf("Discrete (Card)\n");

    printf("\n");
    printf("***********************GPU DEVICE MEMORY INFORMATION***************************\n");
    printf("===============================================================================\n");
    printf("GPU Device Total Memory                                                    : %.0f GB = %.0f MB = %llu Bytes\n", ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f, (float)dev_prop.totalGlobalMem / 1048576.0f, (unsigned long long)dev_prop.totalGlobalMem);
    printf("GPU Device Constant Memory                                                 : %lu Bytes\n", (unsigned long)dev_prop.totalConstMem);
    printf("GPU Dvice Shared Memory Per SMProcessor                                    : %lu\n", (unsigned long)dev_prop.sharedMemPerBlock);
    printf("\n");
    printf("******************* GPU DEVICE DRIVER INFORMATION ******************************\n");
    printf("================================================================================\n");
    printf("GPU Device has ECC support                                                 : %s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");
    #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)   
    printf("GPU Device CUDA Driver Mode (TCC or WDDM)                                  : %s\n", dev_prop.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
    #endif
    printf("***************************************************************************************************************************************************************************************************************************************************************\n");
   
   }
  }
}