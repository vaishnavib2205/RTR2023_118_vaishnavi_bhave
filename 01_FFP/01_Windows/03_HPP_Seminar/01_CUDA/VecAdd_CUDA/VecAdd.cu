#include "hip/hip_runtime.h"
// header file
// Standard headers
#include <stdio.h>

// CUDA
#include<hip/hip_runtime.h>
#include "helper_timer.h"

// global variables
//const int iNumberOfArrayElements = 5;
const int iNumberOfArrayElements = 11444777;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;
float* gold = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

// CUDA kernel
__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
	// code
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i<len)
	{
		out[i] = in1[i] + in2[i];
	}
}

// Entry-point function
int main(void)
{
	// local function declaration
	void fillFloatArrayWithRandomNumbers(float* ,int);
	void vecAddCPU(const float*,const float*,float*,int);
	void cleanup(void);

	// Variable declaration
	int size = iNumberOfArrayElements * sizeof(float);
	hipError_t result = hipSuccess;

	// code
	// host memory allocation
	hostInput1 = (float*)malloc(size);
	if(hostInput1 == NULL)
	{
		printf("Host Memory allocation is failed for hostInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float*)malloc(size);
	if(hostInput2 == NULL)
	{
		printf("Host Memory allocation is failed for hostInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostOutput = (float*)malloc(size);
	if(hostOutput == NULL)
	{
		printf("Host Memory allocation is failed for hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (float*)malloc(size);
	if(gold == NULL)
	{
		printf("Host Memory allocation is failed for gold array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	// filling values into host arrays
	fillFloatArrayWithRandomNumbers(hostInput1,iNumberOfArrayElements);
	fillFloatArrayWithRandomNumbers(hostInput2,iNumberOfArrayElements);

	// device memory allocation
	result = hipMalloc((void**)&deviceInput1, size);
	if(result != hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceInput2, size);
	if(result != hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceOutput, size);
	if(result != hipSuccess)
	{
		printf("Device Memory allocation is failed for deviceOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// copy data from array into device arrays
	result = hipMemcpy(deviceInput1,hostInput1,size,hipMemcpyHostToDevice);
	if(result != hipSuccess)
	{
		printf("Host to device data copy is failed for deviceInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(deviceInput2,hostInput2,size,hipMemcpyHostToDevice);
	if(result != hipSuccess)
	{
		printf("Host to device data copy is failed for deviceInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// CUDA kernel configuration
	dim3 dimGrid = dim3((int)ceil((float)iNumberOfArrayElements / 256.0f),1,1);
	dim3 dimBlock = dim3(256,1,1);

	// CUDA kernel for Vector Addition
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	vecAddGPU <<< dimGrid,dimBlock >>> (deviceInput1,deviceInput2,deviceOutput,iNumberOfArrayElements);

	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;

	// copy data from device array into host array
	result = hipMemcpy(hostOutput , deviceOutput,size,hipMemcpyDeviceToHost);
	if(result != hipSuccess)
	{
		printf("Device to host data copy is failed for hostOutput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	// Vector addition on host
	vecAddCPU(hostInput1,hostInput2,gold,iNumberOfArrayElements);

	// comparison
	const float epsilon = 0.000001f;
	int breakValue = -1;
	bool bAccuracy = true;
	for(int i = 0; i < iNumberOfArrayElements;i++)
	{
		float val1 = gold[i];
		float val2 = hostOutput[i];
		if(fabs(val1-val2)>epsilon)
		{
			bAccuracy = false;
			breakValue = i;
			break;
		}
	}

	char str[128];
	if(bAccuracy == false)
	sprintf(str,"Comparison of CPU and GPU Vector Addition is not within accuracy of 0.000001 at index %d",breakValue);

	else
	sprintf(str,"Comparison of CPU and GPU Vector Addition is within accuracy of 0.000001");

	// output
	printf("Array1 begins from 0th index %.6f to %dth index %.6f\n",hostInput1[0],
	iNumberOfArrayElements - 1,hostInput1[iNumberOfArrayElements - 1]);

	printf("Array2 begins from 0th index %.6f to %dth index %.6f\n",hostInput2[0],
	iNumberOfArrayElements - 1,hostInput2[iNumberOfArrayElements - 1]);

	printf("CUDA kernel Grid dimension = %d,%d,%d and Block dimension = %d,%d,%d \n" , dimGrid.x , dimGrid.y , dimGrid.z , dimBlock.x , dimBlock.y , dimBlock.z);

	printf("Output Array begins from 0th index %.6f to %dth index %.6f\n",hostOutput[0],
	iNumberOfArrayElements - 1,hostOutput[iNumberOfArrayElements - 1]);

	printf("Time taken for Vector addition on CPU = %.6f\n",timeOnCPU);
	printf("Time taken for Vector addition on GPU = %.6f\n",timeOnGPU);
	printf("%s\n",str);

	// cleanup
	cleanup();

	return(0);
}

void fillFloatArrayWithRandomNumbers(float* arr, int len)
{
	// code
	const float fscale = 1.0f / (float)RAND_MAX;
	for(int i = 0; i < len; i++)
	{
		arr[i] = fscale * rand();
	}
}

void vecAddCPU(const float* arr1,const float* arr2,float *out,int len)
{
	// code
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for(int i = 0; i < len; i++)
	{
		out[i] = arr1[i] + arr2[i];
	}

	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;
}

void cleanup(void)
{
	// code
	if(deviceOutput)
	{
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	if(deviceInput2)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if(deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	if(gold)
	{
		free(gold);
		gold = NULL;
	}
	
	if(hostOutput)
	{
		free(hostOutput);
		hostOutput = NULL;
	}

	if(hostInput2)
	{
		free(hostInput2);
		hostInput2 = NULL;
	}

	if(hostInput1)
	{
		free(hostInput1);
		hostInput1 = NULL;
	}
}