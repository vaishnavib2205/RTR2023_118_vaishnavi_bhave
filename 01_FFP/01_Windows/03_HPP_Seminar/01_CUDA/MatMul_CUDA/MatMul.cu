#include "hip/hip_runtime.h"
// header files
// standard headers

#include<stdio.h>

// cuda headers

#include<hip/hip_runtime.h>
#include"helper_timer.h"

// macros

#define BLOCK_WIDTH 64

// global variables 

int *hostA = NULL;
int *hostB = NULL;
int *hostC = NULL;
int *gold = NULL;

int *deviceA = NULL;
int *deviceB = NULL;
int *deviceC = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

__global__ void matMulGPU(int* A , int* B , int* C , int numARows , int numAColumns , int numBColumns , int numCColumns)
{
    // code
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < numARows) && (column < numBColumns))
    {
        int value = 0.0f;
        for(int k = 0; k < numAColumns ; k++)
        {
            int a = A[row * numAColumns + k];
            int b = B[k * numBColumns + column];
            value += a*b;
        }
        C[row * numCColumns + column] = value;

    }
}

int main(int argc , char *argv[])
{
    // function declaration

    void InitA(int *data , int  , int);
    void InitB(int *data , int  , int);
    void matMulCPU(int* , int* , int* , int , int , int , int);
    void cleanup();

    // variable declaration

    int numARows = BLOCK_WIDTH;
    int numAColumns = BLOCK_WIDTH;
    int numBRows = BLOCK_WIDTH;
    int numBColumns = BLOCK_WIDTH;

    int numCRows = numARows;
    int numCColumns = numBColumns;

    int numGoldRows = numARows;
    int numGoldColumns = numBColumns;

    int sizeA = numARows * numAColumns * sizeof(int);
    int sizeB = numBRows * numBColumns * sizeof(int);
    int sizeC = numCRows * numCColumns * sizeof(int);
    int sizeGold = numGoldRows * numGoldColumns * sizeof(int);

    hipError_t result = hipSuccess;

    // code 
    // host memory allocation

    hostA = (int*)malloc(sizeA);
    if(hostA == NULL)
    {
        printf("Host Memory allocation is failed for hostA array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    hostB = (int*)malloc(sizeB);
    if(hostB == NULL)
    {
        printf("Host Memory allocation is failed for hostB array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    hostC = (int*)malloc(sizeC);
    if(hostC == NULL)
    {
        printf("Host Memory allocation is failed for hostC array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    gold = (int*)malloc(sizeGold);
    if(gold == NULL)
    {
        printf("Host Memory allocation is failed for gold array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    // printing matrix dimensions and sizes
    printf("The Dimensions Of Matrix 'hostA' Are : %d x %d :\n", numARows , numAColumns);
    printf("The Dimensions Of Matrix 'hostB' Are : %d x %d :\n", numBRows , numBColumns);
    printf("The Dimensions Of Matrix 'hostC' Are : %d x %d :\n", numCRows , numCColumns);
    printf("The Dimensions Of Matrix 'gold' Are : %d x %d :\n", numGoldRows , numGoldColumns);

    printf("Size Of Matrix hostA = %d \n", sizeA);
    printf("Size Of Matrix hostB = %d \n", sizeB);
    printf("Size Of Matrix hostC = %d \n", sizeC);
    printf("Size Of Matrix gold = %d \n", sizeGold);

    // fill source matrices

    InitA(hostA , numARows , numAColumns);
    InitB(hostB , numBRows , numBColumns);

    // device memory allocation

    result = hipMalloc((void**)&deviceA , sizeA);
    if(result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceA array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    result = hipMalloc((void**)&deviceB , sizeB);
    if(result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceB array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    result = hipMalloc((void**)&deviceC , sizeC);
    if(result != hipSuccess)
    {
        printf("Device Memory allocation is failed for deviceC array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    // copy data from host matrices into device matrices

    result = hipMemcpy(deviceA , hostA, sizeA , hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Device to host data copy is faild for deviceA matrix \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceB , hostB, sizeB , hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Device to host data copy is faild for deviceB matrix \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // cuda kernel configration

    dim3 dimGrid = dim3(ceil((int)numBColumns/(int)BLOCK_WIDTH) ,ceil((int)numARows/(int)BLOCK_WIDTH) ,1);
    dim3 dimBlock = dim3(BLOCK_WIDTH , BLOCK_WIDTH , 1);

    // CUDA Kernel for Matrix MultiPlication

    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

	matMulGPU <<<dimGrid, dimBlock >>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBColumns,numCColumns);

    sdkStopTimer(&timer);
    timeOnGPU = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    timer = NULL;

    // copy data from device matrix into host matrix

    result = hipMemcpy(hostC , deviceC , sizeC , hipMemcpyDeviceToHost);
    if(result != hipSuccess)
    {
        printf("Device To Host Data Copy is failed for hostC matrix \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    // matrix multiplication on host

    matMulCPU(hostA , hostB , hostC , numARows , numAColumns , numBColumns , numCColumns);

    // comparison

    int breakValue = 1;
    bool bAccuracy = true;
    for(int i = 0; i<numCRows * numCColumns; i++)
    {
        int val1 = gold[i];
        int val2 = hostC[i];
        if(val1 != val2)
        {
            bAccuracy = false;
            breakValue = i;
            break;
        }
    }

    char str[128];
    if(bAccuracy == false)
    {
        sprintf(str , "Comparision of CPU and GPU Matrix Multiplication is not accurate at array index %d", breakValue);
    }
    else
    {
        sprintf(str , "Comparision of CPU and GPU Matrix Multiplication is accurate");
    }

    printf("Time Taken for Matrix Multiplication on CPU = %.6f \n" , timeOnCPU);
    printf("Time Taken for Matrix Multiplication on GPU = %.6f \n" , timeOnGPU);
    printf("%s\n" , str);   

    // cleanup 

    cleanup();

    return 0;

}

void InitA(int *data , int row , int col)
{
    int num = 1;
    //code

    for(int i = 0; i<row;i++)
    {
        for(int j = 0; j<col;j++)
        {
            *(data + i * col + j) = num;
            num++;
        }
    }
}

void InitB(int *data , int row , int col)
{
    int num = BLOCK_WIDTH;
    //code

    for(int i = 0; i<row;i++)
    {
        for(int j = 0; j<col;j++)
        {
            *(data + i * col + j) = num;
            num--;
        }
    }
}


void matMulCPU(int* A , int* B , int* C , int numARows , int numAColumns , int numBColumns , int numCColumns)
{
    // code
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for(int i = 0; i<numARows;i++)
    {
        for(int j = 0; j<numBColumns;j++)
        {
            
                int value = 0.0f;
                for(int k = 0; k<numAColumns;k++)
                {
                    int a = A[i * numAColumns + k];
                    int b = B[i * numBColumns + j];
                    value += a*b;
                }
                C[i * numCColumns + j] = value;

            }
        }

        sdkStopTimer(&timer);
        timeOnCPU = sdkGetTimerValue(&timer);
        sdkDeleteTimer(&timer);
        timer = NULL;
}

void cleanup(void)
{
    // code
    if(deviceC)
    {
        hipFree(deviceC);
        deviceC = NULL;
    }

    if(deviceB)
    {
        hipFree(deviceB);
        deviceB = NULL;
    }

    if(deviceA)
    {
        hipFree(deviceA);
        deviceA = NULL;
    }

    if(gold)
    {
        free(gold);
        gold = NULL;
    }

    if(hostC)
    {
        free(hostC);
        hostC = NULL;
    }
    if(hostB)
    {
        free(hostB);
        hostB = NULL;
    }
    if(hostA)
    {
        free(hostA);
        hostA = NULL;
    }



}