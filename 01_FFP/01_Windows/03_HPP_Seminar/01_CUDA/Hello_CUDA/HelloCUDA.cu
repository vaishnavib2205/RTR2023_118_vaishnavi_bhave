// header files
// standard headers

#include<stdio.h>

// cuda headers

#include<hip/hip_runtime.h>

// global variables

const int iNumberOfArrayElements = 5;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

// cuda kernel

__global__ void vecAddGPU(float* in1 , float* in2 , float* out , int len)
{
    // code
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i<len)
    {
        out[i] = in1[i] + in2[i];
    }

}

// entry point function

int main(void)
{
    // function declaration
    void cleanup(void);

    // variable declaration

    int size = iNumberOfArrayElements * sizeof(float);
    hipError_t result = hipSuccess;

    // code
    // host memory allocation
    hostInput1 = (float*)malloc(size);
    if(hostInput1 == NULL)
    {
        printf("Host Memory allocation is failed for hostInput1 array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    hostInput1 = (float*)malloc(size);
    if(hostInput1 == NULL)
    {
        printf("Host Memory allocation is failed for hostInput1 array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    hostInput2 = (float*)malloc(size);
    if(hostInput2 == NULL)
    {
        printf("Host Memory allocation is failed for hostInput2 array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    hostOutput = (float*)malloc(size);
    if(hostOutput == NULL)
    {
        printf("Host Memory allocation is failed for hostOutput array. \n");
        cleanup();
        exit(EXIT_FAILURE); 
    }

    // filling values into host arrays

    hostInput1[0] = 101.0;
    hostInput1[1] = 102.0;
    hostInput1[2] = 103.0;
    hostInput1[3] = 104.0;
    hostInput1[4] = 105.0;

    hostInput2[0] = 201.0;
    hostInput2[1] = 202.0;
    hostInput2[2] = 203.0;
    hostInput2[3] = 204.0;
    hostInput2[4] = 205.0;

    // device memory allocation

    result = hipMalloc((void**)&deviceInput1 ,size);
    if(result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceInput1 array. ");
    }

    result = hipMalloc((void**)&deviceInput2 ,size);
    if(result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceInput2 array. ");
    }

    result = hipMalloc((void**)&deviceOutput ,size);
    if(result != hipSuccess)
    {
        printf("Device memory allocation is failed for deviceOutput array. ");
    }

    // copy data from host arrays into device arrays

    result = hipMemcpy(deviceInput1 , hostInput1 , size , hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to device data copy is faild for deviceInput1 array \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    result = hipMemcpy(deviceInput2 , hostInput2 , size , hipMemcpyHostToDevice);
    if(result != hipSuccess)
    {
        printf("Host to device data copy is faild for deviceInput2 array \n");
        cleanup();
        exit(EXIT_FAILURE);
    }

    dim3 dimGrid = dim3(iNumberOfArrayElements , 1, 1);
    dim3 dimBlock = dim3(1,1,1);

    // CUDA kernel for vector Addition
    vecAddGPU <<<dimGrid , dimBlock>>> (deviceInput1 , deviceInput2 , deviceOutput , iNumberOfArrayElements);

    // copy data from array into host array

    result = hipMemcpy(hostOutput , deviceOutput , size , hipMemcpyDeviceToHost);
    if(result != hipSuccess)
    {
        printf("Device To Host Data Copy is failed for hostOutput array \n");
        cleanup();
        exit(EXIT_FAILURE);
    }


    // vector addition on host

    for(int i = 0; i<iNumberOfArrayElements; i++)
    {
        printf("%f + %f = %f\n" , hostInput1[i] , hostInput2[i] , hostOutput[i]);
    }

    cleanup();

    return 0;
}

void cleanup(void)
{
    // code

    if(deviceOutput)
    {
        hipFree(deviceOutput);
        deviceOutput = NULL;
    }

    if(deviceInput2)
    {
        hipFree(deviceInput2);
        deviceInput2 = NULL;
    }

    if(deviceInput1)
    {
        hipFree(deviceInput1);
        deviceInput1 = NULL;
    }

    if(hostOutput)
    {
        hipFree(hostOutput);
        hostOutput = NULL;
    }

    if(hostInput2)
    {
        hipFree(hostInput2);
        hostInput2 = NULL;
    }

    if(hostInput1)
    {
        hipFree(hostInput1);
        hostInput1 = NULL;
    }

}